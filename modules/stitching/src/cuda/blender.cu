#include "hip/hip_runtime.h"
#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/cudev/ptr2d/glob.hpp"
#include "opencv2/cudev/grid/transform.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"

using namespace cv::cudev;

namespace cv { namespace cuda { namespace device {

template <typename T>
__global__ void do_vr_add_sub_and_multiply(const GlobPtr<T> a,
                                           const GlobPtr<T> t,
                                           const GlobPtr<float> w,
                                           GlobPtr<short3> d,
                                           const int rows, const int cols) {

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < cols && y < rows) {
            short3 sub;
            T a_elem = a.row(y)[x];
            T t_elem = t.row(y)[x];
            float w_elem = w.row(y)[x];

            sub.x = (a_elem.x - t_elem.x) * w_elem;
            sub.y = (a_elem.y - t_elem.y) * w_elem;
            sub.z = (a_elem.z - t_elem.z) * w_elem;

            short3 * d_p = d.row(y) + x;
            (*d_p).x += sub.x;
            (*d_p).y += sub.y;
            (*d_p).z += sub.z;
        }
}

// used by MultiBandGPUBlender
// D += (A - T) * W
template <typename TYPE>
__host__ void vr_add_sub_and_multiply(const GpuMat & A, 
                                      const GpuMat & T, 
                                      const GpuMat & W, 
                                      GpuMat & D, hipStream_t stream) {
    CV_Assert(A.type() == CV_8UC3 || A.type() == CV_8UC4);
    CV_Assert(T.type() == CV_8UC3 || T.type() == CV_8UC4);
    CV_Assert(W.type() == CV_32F);
    CV_Assert(D.type() == CV_16SC3);
    CV_Assert(A.size() == T.size() && A.size() == W.size() && A.size() == D.size());

    const dim3 block(DefaultTransformPolicy::block_size_x, DefaultTransformPolicy::block_size_y);
    const dim3 grid(divUp(A.cols, block.x), divUp(A.rows, block.y));

    do_vr_add_sub_and_multiply<<<grid, block, 0 stream>>>(globPtr<TYPE>(A),
                                                          globPtr<TYPE>(T),
                                                          globPtr<float>(W),
                                                          globPtr<short3>(D),
                                                          A.rows, A.cols);
    CV_CUDEV_SAFE_CALL( hipGetLastError() );
    CV_CUDEV_SAFE_CALL( hipDeviceSynchronize() );
}

template void vr_add_sub_and_multiply<uchar3>(const GpuMat &, const GpuMat &, const GpuMat &, GpuMat &, hipStream_t);
template void vr_add_sub_and_multiply<uchar4>(const GpuMat &, const GpuMat &, const GpuMat &, GpuMat &, hipStream_t);

template <typename T>
__global__ void do_vr_add_multiply(const GlobPtr<T> a,
                                   const GlobPtr<float> w,
                                   GlobPtr<short3> d,
                                   const int rows, const int cols) {

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < cols && y < rows) {
            short3 sub;
            T a_elem = a.row(y)[x];
            float w_elem = w.row(y)[x];

            sub.x = (a_elem.x) * w_elem;
            sub.y = (a_elem.y) * w_elem;
            sub.z = (a_elem.z) * w_elem;

            short3 * d_p = d.row(y) + x;
            (*d_p).x += sub.x;
            (*d_p).y += sub.y;
            (*d_p).z += sub.z;
        }
}

template <typename TYPE>
__host__ void vr_add_multiply(const GpuMat & A, 
                              const GpuMat & W, 
                              GpuMat & D, hipStream_t stream) {
    CV_Assert(A.type() == CV_8UC3 || A.type() == CV_8UC4);
    CV_Assert(W.type() == CV_32F);
    CV_Assert(D.type() == CV_16SC3);
    CV_Assert(A.size() == W.size() && A.size() == D.size());

    const dim3 block(DefaultTransformPolicy::block_size_x, DefaultTransformPolicy::block_size_y);
    const dim3 grid(divUp(A.cols, block.x), divUp(A.rows, block.y));

    do_vr_add_multiply<<<grid, block, 0 stream>>>(globPtr<TYPE>(A),
                                                  globPtr<float>(W),
                                                  globPtr<short3>(D),
                                                  A.rows, A.cols);
    CV_CUDEV_SAFE_CALL( hipGetLastError() );
    CV_CUDEV_SAFE_CALL( hipDeviceSynchronize() );
}

template void vr_add_multiply<uchar3>(const GpuMat &, const GpuMat &, GpuMat &, hipStream_t);
template void vr_add_multiply<uchar4>(const GpuMat &, const GpuMat &, GpuMat &, hipStream_t);

}}} // namespace cv { namespace cuda { namespace cudev {


#endif /* CUDA_DISABLER */
